#include "hip/hip_runtime.h"

// This is not really C++-code but pretty plain C code, but we compile it
// as C++ so we can integrate with CUDA seamlessly.

// If you plan on submitting your solution for the Parallel Sorting Contest,
// please keep the split into main file and kernel file, so we can easily
// insert other data.
const int BLOCKSIZE = 1024;

__device__
static void exchange(int *i, int *j)
{
	int k;
	k = *i;
	*i = *j;
	*j = k;
}

__global__ void bitonic_sort(int *data, int N, int j, int k) {
		int i =  threadIdx.x + blockDim.x*blockIdx.x;
		if (i < N){

			int ixj=i^j; // Calculate indexing!
			if ((ixj)>i)
			{
				if ((i&k)==0 && data[i]>data[ixj]) exchange(&data[i],&data[ixj]);
				if ((i&k)!=0 && data[i]<data[ixj]) exchange(&data[i],&data[ixj]);
			}
		}
}



// No, this is not GPU code yet but just a copy of the CPU code, but this
// is where I want to see your GPU code!
void bitonic_gpu(int *data, int N)
{
  int i,j,k;
	int size = sizeof(int) * N;
	int *gpudata;
	hipMalloc( (void**)&gpudata, size);
	hipMemcpy(gpudata, data, size, hipMemcpyHostToDevice );
	dim3 dimBlock( BLOCKSIZE, 1 );
	dim3 dimGrid( N/BLOCKSIZE,1 );


  for (k=2;k<=N;k=2*k) // Outer loop, double size for each step
  {
    for (j=k>>1;j>0;j=j>>1) // Inner loop, half size for each step
    {
			bitonic_sort<<<dimGrid, dimBlock>>>(gpudata, N, j, k);
			hipDeviceSynchronize();
    }
  }

	hipMemcpy(data, gpudata, size, hipMemcpyDeviceToHost );
}
