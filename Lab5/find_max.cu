#include "hip/hip_runtime.h"
// Reduction lab, find maximum

#include <stdio.h>
#include "milli.h"

const int BLOCKSIZE = 1024;
__global__ void find_max(int *indata, int *outdata, int N)
{

  __shared__ int sdata[BLOCKSIZE];
  int i;
  i = threadIdx.x + blockDim.x*blockIdx.x;
  int tid = threadIdx.x;
  sdata[tid] = (i < N ? indata[i] : -1);

  __syncthreads();

  for (unsigned int s = blockDim.x/2; s > 0; s>>=1){
    if (tid < s){
      sdata[tid] = (sdata[tid] < sdata[tid+s] ? sdata[tid+s] : sdata[tid]);
    }
    __syncthreads();
  }

  if (tid==0)
    outdata[blockIdx.x] = sdata[0];

}

void launch_cuda_kernel(int *data, int N)
{
	// Handle your CUDA kernel launches in this function

	int *devdata;
  int *outdata;
	int size = sizeof(int) * N;
	hipMalloc( (void**)&devdata, size);
	hipMemcpy(devdata, data, size, hipMemcpyHostToDevice );
	hipMalloc( (void**)&outdata, size);
	// Dummy launch
  int currentCount = N;
  while(currentCount > 1){
  	dim3 dimBlock( BLOCKSIZE, 1 );
  	dim3 dimGrid( (currentCount/BLOCKSIZE)+1, 1 );
  	find_max<<<dimGrid, dimBlock>>>(devdata, outdata, currentCount);

  	//hipMemcpy(data, outdata, sizeof(int)*currentCount/2, hipMemcpyDeviceToHost );
    /*
    for (int i = 0; i < currentCount/2; i ++){
      printf("%d ,", data[i]);
    }
    printf("\n");*/
    int * tmp = outdata;
    outdata = devdata;
    devdata = tmp;

    hipError_t err = hipPeekAtLastError();
  	if (err) printf("hipPeekAtLastError %d %s\n", err, hipGetErrorString(err));


    currentCount /= BLOCKSIZE;
  }
	// Only the result needs copying!

	hipMemcpy(data, devdata, sizeof(int), hipMemcpyDeviceToHost );
	hipFree(devdata);
  hipFree(outdata);
}

// CPU max finder (sequential)
void find_max_cpu(int *data, int N)
{
  int i, m;

	m = data[0];
	for (i=0;i<N;i++) // Loop over data
	{
		if (data[i] > m)
			m = data[i];
	}
	data[0] = m;
}

#define SIZE 50000000
//#define SIZE 16
// Dummy data in comments below for testing
int data[SIZE];// = {1, 2, 5, 3, 6, 8, 5, 3, 1, 65, 8, 5, 3, 34, 2, 54};
int data2[SIZE];// = {1, 2, 5, 3, 6, 8, 5, 3, 1, 65, 8, 5, 3, 34, 2, 54};
//int data[] = {1, 2, 5, 3, 6, 8, 5, 3, 1, 65, 8, 5, 3, 34, 2, 54};
//int data2[] = {1, 2, 5, 3, 6, 8, 5, 3, 1, 65, 8, 5, 3, 34, 2, 54};

int main()
{
  // Generate 2 copies of random data

  srand(time(NULL));
  for (long i=0;i<SIZE;i++)
  {
    data[i] = rand() % (SIZE * 5);
    data2[i] = data[i];
  }

  // The GPU will not easily beat the CPU here!
  // Reduction needs optimizing or it will be slow.
  ResetMilli();
  find_max_cpu(data, SIZE);
  printf("CPU time %f\n", GetSeconds());
  ResetMilli();
  launch_cuda_kernel(data2, SIZE);
  printf("GPU time %f\n", GetSeconds());

  // Print result
  printf("\n");
  printf("CPU found max %d\n", data[0]);
  printf("GPU found max %d\n", data2[0]);
}
