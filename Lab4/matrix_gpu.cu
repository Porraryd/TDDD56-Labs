#include "hip/hip_runtime.h"

#include <stdio.h>

const int N = 1024;

const int blocksize = 128;


__global__
void matrix_add(float *a, float *b, float* c)
{

  int y = (blockIdx.x * blockDim.x + threadIdx.x);
  int x = (blockIdx.y * blockDim.y + threadIdx.y);
  int offset = x+ (y * 1024;
  //printf("x: %d, y: %d \n", x, y);
	c[offset] = a[offset]+b[offset];
}

int main()
{

	const int size = N*N*sizeof(float);

  float* a = new float[N*N];
	float* b = new float[N*N];
	float* result = new float[N*N];
  float theTime = 0;

	float* c;
	float* aa;
	float* bb;

  for (int i = 0; i < N; i++)
		for (int j = 0; j < N; j++)
		{
			a[i+j*N] = 10 + i;
			b[i+j*N] = (float)j / N;
		}

hipError_t err;

  hipEvent_t myEvent;
  hipEvent_t mySecondEvent;
  hipEventCreate(&myEvent);
  hipEventSynchronize(myEvent);
  hipEventCreate(&mySecondEvent);
	hipMalloc( (void**)&c, size );
	hipMalloc( (void**)&aa, size );
	hipMalloc( (void**)&bb, size );

	hipMemcpy(aa, a, size, hipMemcpyHostToDevice);
  hipMemcpy(bb, b, size, hipMemcpyHostToDevice);

	dim3 dimBlock( 16, 16 );
	dim3 dimGrid( 64, 64 );

  hipEventRecord(myEvent, 0);
	matrix_add<<<dimGrid, dimBlock>>>(aa,bb,c);
  hipEventRecord(mySecondEvent, 0);
  hipEventSynchronize(mySecondEvent);
  hipEventElapsedTime(&theTime, myEvent, mySecondEvent);
  printf("Elapsed time: %f \n", theTime/1000);
  err = hipPeekAtLastError();
  if(err) printf("hipPeekAtLastError %d %s\n", err, hipGetErrorString(err));


	hipDeviceSynchronize();
	hipMemcpy( result, c, size, hipMemcpyDeviceToHost );
	hipFree( c );
  hipFree( aa );
  hipFree( bb );

/*
for (int i = 0; i < N; i++)
{
  for (int j = 0; j < N; j++)
  {
    printf("%0.2f ", result[i+j*N]);
  }
  printf("\n");
}*/
	printf("done\n");
	return EXIT_SUCCESS;
}
