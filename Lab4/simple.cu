#include "hip/hip_runtime.h"
// Simple CUDA example by Ingemar Ragnemalm 2009. Simplest possible?
// Assigns every element in an array with its index.

// nvcc simple.cu -L /usr/local/cuda/lib -lcudart -o simple

#include <stdio.h>

const int N = 16;
const int blocksize = 16;
const float input[16] = { 1.f, 2.f, 4.f, 8.f,
									16.f, 32.f, 64.f, 128.f,
								 256.f, 512.f, 1024.f, 2048.f,
								4096.f, 8192.f, 16284.f, 32568.f };

__global__
void simple(float *c)
{
	c[threadIdx.x] = sqrtf(c[threadIdx.x]);
}

int main()
{
	float *c = new float[N];
	float *cd;
	const int size = N*sizeof(float);
	hipMalloc( (void**)&cd, size );

	hipMemcpy(cd, (void*)input, size, hipMemcpyHostToDevice);
	dim3 dimBlock( blocksize, 1 );
	dim3 dimGrid( 1, 1 );
	simple<<<dimGrid, dimBlock>>>(cd);
	hipDeviceSynchronize();
	hipMemcpy( c, cd, size, hipMemcpyDeviceToHost );
	hipFree( cd );

	for (int i = 0; i < N; i++)
		printf("%f ", c[i]);
	printf("\n");

	for (int i = 0; i < N; i++)
		printf("%f ", sqrtf(input[i]));

	delete[] c;
	printf("done\n");
	return EXIT_SUCCESS;
}
